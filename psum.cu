
#include <hip/hip_runtime.h>
#include <stdio.h>
#define DSIZE 512
#define cudaCheckErrors(msg) \
    do { \
        hipError_t __err = hipGetLastError(); \
        if (__err != hipSuccess) { \
            fprintf(stderr, "Fatal error: %s (%s at %s:%d)\n", \
                msg, hipGetErrorString(__err), \
                __FILE__, __LINE__); \
            fprintf(stderr, "*** FAILED - ABORTING\n"); \
            exit(1); \
        } \
    } while (0)


typedef int mytype;

template <typename T>
__global__ void prescan(T *g_odata, T *g_idata, int n)
{
  extern __shared__ T temp[];  // allocated on invocation
  int thid = threadIdx.x;
  int offset = 1;
  temp[2*thid] = g_idata[2*thid]; // load input into shared memory
  temp[2*thid+1] = g_idata[2*thid+1];
  for (int d = n>>1; d > 0; d >>= 1)                    // build sum in place up the tree
  {
    __syncthreads();
    if (thid < d)
    {
      int ai = offset*(2*thid+1)-1;
      int bi = offset*(2*thid+2)-1;
      temp[bi] += temp[ai];
    }
    offset *= 2;
  }
  if (thid == 0) { temp[n - 1] = 0; } // clear the last element
  for (int d = 1; d < n; d *= 2) // traverse down tree & build scan
    {
      offset >>= 1;
      __syncthreads();
      if (thid < d)
      {
         int ai = offset*(2*thid+1)-1;
         int bi = offset*(2*thid+2)-1;
         T t = temp[ai];
         temp[ai] = temp[bi];
         temp[bi] += t;
      }
    }
  __syncthreads();
  g_odata[2*thid] = temp[2*thid]; // write results to device memory
  g_odata[2*thid+1] = temp[2*thid+1];
}

int main(){

  mytype *h_i, *d_i, *h_o, *d_o;
  int dszp = (DSIZE)*sizeof(mytype);

  h_i = (mytype *)malloc(dszp);
  h_o = (mytype *)malloc(dszp);
  if ((h_i == NULL) || (h_o == NULL)) {printf("malloc fail\n"); return 1;}
  hipMalloc(&d_i, dszp);
  hipMalloc(&d_o, dszp);
  cudaCheckErrors("hipMalloc fail");
  for (int i = 0 ; i < DSIZE; i++){
    h_i[i] = i;
    h_o[i] = 0;}
  hipMemset(d_o, 0, dszp);
  cudaCheckErrors("hipMemset fail");
  hipMemcpy(d_i, h_i, dszp, hipMemcpyHostToDevice);
  cudaCheckErrors("hipMemcpy 1 fail");
  prescan<<<1,DSIZE/2, dszp>>>(d_o, d_i, DSIZE);
  hipDeviceSynchronize();
  cudaCheckErrors("kernel fail");
  hipMemcpy(h_o, d_o, dszp, hipMemcpyDeviceToHost);
  cudaCheckErrors("hipMemcpy 2 fail");
  mytype psum = 0;
  for (int i =1; i < DSIZE; i++){
    psum += h_i[i-1];
    if (psum != h_o[i]) {printf("mismatch at %d, was: %d, should be: %d\n", i, h_o[i], psum); return 1;}
    }


    //print input and output
    printf("The input of the program is: \n");
    for(int i = 0; i < DSIZE && i < 20; i++){
        printf("%d\n",h_i[i]);
    }

    printf("The output of the program is: \n");
    for(int i = 0; i < DSIZE && i < 20; i++){
        printf("%d\n",h_o[i]);
    }
  return 0;
}