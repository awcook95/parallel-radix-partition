#include "hip/hip_runtime.h"
#include <assert.h>
#include <stdio.h>
#include <math.h>

#define RAND_RANGE(N) ((double)rand()/((double)RAND_MAX + 1)*(N))

//data generator
void dataGenerator(int* data, int count, int first, int step)
{
	assert(data != NULL);

	for(int i = 0; i < count; ++i)
		data[i] = first + i * step;
	srand(time(NULL));
    for(int i = count-1; i>0; i--) //knuth shuffle
    {
        int j = RAND_RANGE(i);
        int k_tmp = data[i];
        data[i] = data[j];
        data[j] = k_tmp;
    }
}

//non random data generator
void nrdataGenerator(int* data, int count, int first, int step)
{
	assert(data != NULL);

	for(int i = 0; i < count; ++i)
		data[i] = first + i * step;
	//srand(time(NULL));
    for(int i = count-1; i>0; i--) //knuth shuffle
    {
        int j = RAND_RANGE(i);
        int k_tmp = data[i];
        data[i] = data[j];
        data[j] = k_tmp;
    }
}

/* This function embeds PTX code of CUDA to extract bit field from x. 
   "start" is the starting bit position relative to the LSB. 
   "nbits" is the bit field length.
   It returns the extracted bit field as an unsigned integer.
*/
__device__ uint bfe(uint x, uint start, uint nbits)
{
    uint bits;
    asm("bfe.u32 %0, %1, %2, %3;" : "=r"(bits) : "r"(x), "r"(start), "r"(nbits));
    return bits;
}

//Feel free to change the names of the kernels or define more kernels below if necessary

//define the histogram kernel here
__global__ void histogram(int* d_data, int* d_histogram, int tagLength, int size)
{
    int T = blockIdx.x * blockDim.x + threadIdx.x;

    if(T < size){
        int h = bfe(d_data[T], 0, tagLength);
        atomicAdd(&(d_histogram[h]), 1);
    }
}

//define the prefix scan kernel here
//implement it yourself or borrow the code from CUDA samples
__global__ void prefixScan(int* d_histogram, int* sum, int size)
{

}

//Cuda sample code - step 1 of exclusive parallel scan
__global__ void shfl_scan_test(int *data, int width, int *partial_sums=NULL)
{
    extern __shared__ int sums[];
    int id = ((blockIdx.x * blockDim.x) + threadIdx.x);
    int lane_id = id % warpSize;
    // determine a warp_id within a block
    int warp_id = threadIdx.x / warpSize;

    // Below is the basic structure of using a shfl instruction
    // for a scan.
    // Record "value" as a variable - we accumulate it along the way
    int value = data[id];

    // Now accumulate in log steps up the chain
    // compute sums, with another thread's value who is
    // distance delta away (i).  Note
    // those threads where the thread 'i' away would have
    // been out of bounds of the warp are unaffected.  This
    // creates the scan sum.
#pragma unroll

    for (int i=1; i<=width; i*=2)
    {
        int n = __shfl_up(value, i, width);

        if (lane_id >= i) value += n;
    }

    // value now holds the scan value for the individual thread
    // next sum the largest values for each warp

    // write the sum of the warp to smem
    if (threadIdx.x % warpSize == warpSize-1)
    {
        sums[warp_id] = value;
    }

    __syncthreads();

    //
    // scan sum the warp sums
    // the same shfl scan operation, but performed on warp sums
    //
    if (warp_id == 0 && lane_id < (blockDim.x / warpSize))
    {
        int warp_sum = sums[lane_id];

        for (int i=1; i<=width; i*=2)
        {
            int n = __shfl_up(warp_sum, i, width);

            if (lane_id >= i) warp_sum += n;
        }

        sums[lane_id] = warp_sum;
    }

    __syncthreads();

    // perform a uniform add across warps in the block
    // read neighbouring warp's sum and add it to threads value
    int blockSum = 0;

    if (warp_id > 0)
    {
        blockSum = sums[warp_id-1];
    }

    value += blockSum;

    // Now write out our result
    data[id] = value;

    // last thread has sum, write write out the block's sum
    if (partial_sums != NULL && threadIdx.x == blockDim.x-1)
    {
        partial_sums[blockIdx.x] = value;
    }
}

//Cuda sample code - step 2 of exclusive parallel scan
__global__ void uniform_add(int *data, int *partial_sums, int len)
{
    __shared__ int buf;
    int id = ((blockIdx.x * blockDim.x) + threadIdx.x);

    if (id > len) return;

    if (threadIdx.x == 0)
    {
        buf = partial_sums[blockIdx.x];
    }

    __syncthreads();
    data[id] += buf;
}

//define the reorder kernel here
__global__ void Reorder(int* d_data, int* d_output, int* sum, int tagLength, int size)
{
    int T = blockIdx.x * blockDim.x + threadIdx.x;

    if(T < size){
        int h = bfe(d_data[T], 0, tagLength); //extract bits from input data
        int offset = atomicAdd(&(sum[h]), 1); //calculate offset
        d_output[offset] = d_data[T]; //use the offset to place input data into correct partition
    }
}

bool isPowerOfTwo(unsigned long x)
{
    return (x != 0) && ((x & (x - 1)) == 0);
}

void outputHistogram(int* histogram, int buckets){
    for(int i = 0; i < buckets; i++){
        printf("\n%02d: ", i);
        printf("%15lld ", histogram[i]);
    }
    printf("\n");
}

static unsigned int iDivUp(unsigned int dividend, unsigned int divisor)
{
    return ((dividend % divisor) == 0) ?
           (dividend / divisor) :
           (dividend / divisor + 1);
}

int main(int argc, char const *argv[])
{
    int rSize = atoi(argv[1]); //number of elements in input array
    int numP = atoi(argv[2]); //number of partitions that input will be sorted into
    
    int* h_data; //input array

    hipHostMalloc((void**)&h_data, sizeof(int)*rSize); //use pinned memory in host so it copies to GPU faster
    
    nrdataGenerator(h_data, rSize, 0, 1); //randomly generate input data
    
    /* your code */


    assert(numP <= rSize && isPowerOfTwo(numP)); //number of partitions must be less than or equal to the input array size and power of 2

    int tag = int(log2(float(numP))); //define number of bits in a tag

    
    printf("The number of elements in the input array is: %d\n",rSize);
    printf("The number of partitions is: %d\n",numP);
    printf("The number of bits in a tag is: %d\n\n",tag);

    printf("The contents of the input array are: \n");
    for(int i = 0; i < rSize && i < 10; i++){
        
        printf("%d\n",h_data[i]);
    }

    //allocate memory for host

    //(input array already allocated above)

    int* h_histogram; //host histogram
    hipHostMalloc((void**)&h_histogram, sizeof(int)*numP); //a bucket for each partition

    memset(h_histogram, 0, sizeof(int)*numP); //initialize host histogram to zero

    //Allocate device memory
    int* d_data; //input array for device
    int* d_histogram; //histogram for device

    hipMalloc((void**)&d_data, sizeof(int)*rSize); //size of number of inputs
    hipMalloc((void**)&d_histogram, sizeof(int)*numP); //size of number of partitions

    //copy host data to device memory
    hipMemcpy(d_data, h_data, sizeof(int)*rSize, hipMemcpyHostToDevice);
    hipMemcpy(d_histogram, h_histogram, sizeof(int)*numP, hipMemcpyHostToDevice);

    

    //start counting time
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
    hipEventRecord(start, 0);
    
    //prepare kernel 1 - creation of the histogram

        //define block and grid size for kernel 1
        int num_threads = 32; //number of threads in a block
        int num_blocks = (rSize + num_threads - 1)/num_threads;

        histogram<<<num_blocks, num_threads>>>(d_data, d_histogram, tag, rSize);

        //copy data from gpu to host
        hipMemcpy(h_histogram, d_histogram, sizeof(int)*numP, hipMemcpyDeviceToHost);

        //print output
        outputHistogram(h_histogram, numP);

    //prepare kernel 2 - exclusive prefix sum of histogram

        //define block/grid size and other needed variables
        int *h_partial_sums, *h_result;
        int *d_partial_sums;
        const int n_elements = rSize; //number of input elements
        int sz = sizeof(int)*n_elements;

        int blockSize = 256;
        int gridSize = n_elements/blockSize;
        int nWarps = blockSize/32;
        int shmem_sz = nWarps * sizeof(int);
        int n_partialSums = n_elements/blockSize;
        int partial_sz = n_partialSums*sizeof(int);

        //allocate memory
        hipMalloc((void **)&d_partial_sums, partial_sz);

        printf("Scan summation for %d elements, %d partial sums\n",
            n_elements, n_elements/blockSize);

        int p_blockSize = min(n_partialSums, blockSize);
        int p_gridSize = iDivUp(n_partialSums, p_blockSize);
        printf("Partial summing %d elements with %d blocks of size %d\n",
            n_partialSums, p_gridSize, p_blockSize);

        //multiple kernel calls to accomplish the prefix sum
        //shfl_scan_test<<<gridSize,blockSize, shmem_sz>>>(d_data, 32, d_partial_sums);
        //shfl_scan_test<<<p_gridSize,p_blockSize, shmem_sz>>>(d_partial_sums,32);
        //uniform_add<<<gridSize-1, blockSize>>>(d_data+blockSize, d_partial_sums, n_elements);

        //copy data from gpu to host
        //hipMemcpy(h_result, d_data, sz, hipMemcpyDeviceToHost);
        //hipMemcpy(h_partial_sums, d_partial_sums, partial_sz, hipMemcpyDeviceToHost);

    //prepare kernel 3 - reorder input array

    //stop counting time
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	float elapsedTime;
    hipEventElapsedTime(&elapsedTime, start, stop);
    
    //report running time
	printf("******** Total Running Time of Kernel = %0.5f ms *******\n", elapsedTime);
	hipEventDestroy(start);
	hipEventDestroy(stop);

    hipHostFree(h_data);
    hipHostFree(h_histogram);

    return 0;
}
