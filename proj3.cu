#include "hip/hip_runtime.h"
#include <assert.h>
#include <stdio.h>
#include <math.h>

#define RAND_RANGE(N) ((double)rand()/((double)RAND_MAX + 1)*(N))

//data generator
void dataGenerator(int* data, int count, int first, int step)
{
	assert(data != NULL);

	for(int i = 0; i < count; ++i)
		data[i] = first + i * step;
	srand(time(NULL));
    for(int i = count-1; i>0; i--) //knuth shuffle
    {
        int j = RAND_RANGE(i);
        int k_tmp = data[i];
        data[i] = data[j];
        data[j] = k_tmp;
    }
}

//non random data generator
void nrdataGenerator(int* data, int count, int first, int step)
{
	assert(data != NULL);

	for(int i = 0; i < count; ++i)
		data[i] = first + i * step;
	//srand(time(NULL));
    for(int i = count-1; i>0; i--) //knuth shuffle
    {
        int j = RAND_RANGE(i);
        int k_tmp = data[i];
        data[i] = data[j];
        data[j] = k_tmp;
    }
}

//sequential data generator
void sDataGenerator(int* data, int count){
    assert(data !=NULL);
    int j = 0;
    for(int i = count-1; i > 0; --i){
        data[j] = i;
        j++;
    }
}

/* This function embeds PTX code of CUDA to extract bit field from x. 
   "start" is the starting bit position relative to the LSB. 
   "nbits" is the bit field length.
   It returns the extracted bit field as an unsigned integer.
*/
__device__ uint bfe(uint x, uint start, uint nbits)
{
    uint bits;
    asm("bfe.u32 %0, %1, %2, %3;" : "=r"(bits) : "r"(x), "r"(start), "r"(nbits));
    return bits;
}

//Feel free to change the names of the kernels or define more kernels below if necessary

//define the histogram kernel here
__global__ void histogram(int* d_data, int* d_histogram, int tagLength, int size, int num_buckets)
{
    extern __shared__ int s_histogram[];

    for(int i = threadIdx.x; i < num_buckets; i += blockDim.x){ //initialize array to 0 in block sized chunks
		s_histogram[i] = 0;
    }
    
    __syncthreads();

    int T = blockIdx.x * blockDim.x + threadIdx.x;

    for(int i = T; i < size; i += blockDim.x * gridDim.x){ //grid stride
        int h = bfe(d_data[i], 0, tagLength);
        atomicAdd(&(s_histogram[h]), 1);
    }

    __syncthreads();

	//reduce shared output into global output
	for(int i = threadIdx.x; i < num_buckets; i += blockDim.x){ //output to global memory in block sized chunks
		atomicAdd(&(d_histogram[i]), s_histogram[i]);
	}
}

//define the prefix scan kernel here
//implement it yourself or borrow the code from CUDA samples
__global__ void prefixscan(int *d_input, int *d_output, int n)
{
    extern __shared__ int shmem[];
    int T = threadIdx.x;
    int offset = 1;

    //there are n/2 threads so each thread must load 2 data points
    shmem[2*T] = d_input[2*T]; // load even indices into shared memory
    shmem[2*T+1] = d_input[2*T+1]; //load odd indices

    for (int d = n>>1; d > 0; d >>= 1) //upsweep, compute partial sums
    {
        __syncthreads();
        if (T < d)
        {
            int ai = offset*(2*T+1)-1;
            int bi = offset*(2*T+2)-1;
            shmem[bi] += shmem[ai];
        }
    offset *= 2;
    }

    if (T == 0) { shmem[n - 1] = 0; } //last element to 0
    for (int d = 1; d < n; d *= 2) //downsweep, use partial sums to complete the psum
    {
        offset >>= 1;
        __syncthreads();

        if (T < d){
         int ai = offset*(2*T+1)-1;
         int bi = offset*(2*T+2)-1;
         int temp = shmem[ai];
         shmem[ai] = shmem[bi];
         shmem[bi] += temp;
        }
    }

    __syncthreads();
    d_output[2*T] = shmem[2*T]; //write to global memory in even odd pairs like above
    d_output[2*T+1] = shmem[2*T+1];
}

//define the reorder kernel here
__global__ void Reorder(int* d_data, int* d_output, int* sum, int tagLength, int size)
{
    int T = blockIdx.x * blockDim.x + threadIdx.x;

    if(T < size){
        int h = bfe(d_data[T], 0, tagLength); //extract bits from input data
        int offset = atomicAdd(&(sum[h]), 1); //calculate offset
        d_output[offset] = d_data[T]; //use the offset to place input data into correct partition
    }
}

bool isPowerOfTwo(unsigned long x)
{
    return (x != 0) && ((x & (x - 1)) == 0);
}

void outputHistogram(int* histogram, int buckets){
    printf("Histogram:");
    for(int i = 0; i < buckets && i < 10; i++){
        printf("\n%02d: ", i);
        printf("%15lld ", histogram[i]);
    }
}

void output_result(int* histogram, int* psum, int num_buckets){
	int i; 
    long long total_cnt = 0;
    printf("Partition number:     Offset and number of keys per partition:");
	for(i=0; i< num_buckets; i++) {
		if(i%5 == 0) /* we print 5 buckets in a row */
			printf("\n%02d: ", i);
		printf("%8d: %7lld ", psum[i],histogram[i]);
		total_cnt += histogram[i];
	  	/* we also want to make sure the total distance count is correct */
		if(i == num_buckets - 1)	
			printf("\n T:%lld \n", total_cnt);
		else printf("| ");
	}
}

int main(int argc, char const *argv[])
{
    int rSize = atoi(argv[1]); //number of elements in input array
    int numP = atoi(argv[2]); //number of partitions that input will be sorted into

    assert(numP <= rSize && isPowerOfTwo(numP)); //number of partitions must be less than or equal to the input array size and power of 2
    int tag = int(log2(float(numP))); //define number of bits in a tag

    //errors for incorrect inputs
    if(argc > 3){
		printf("Too many command line arguments, ending program\n");
		return 0;
	}

	else if(argc < 3){
		printf("Too few command line arguments, ending program\n");
		return 0;
    }
    
    if(rSize <= 0 || numP > 1024 || numP <=0){ //input size must be >= 0 and max # partitions is 1024
		printf("Invalid command line arguments, ending program\n");
		return 0;
	}
    
    //Create all needed arrays for all 3 kernels and allocate memory

    int* r_d; //input array for device
    hipMalloc((void**)&r_d, sizeof(int)*rSize); //size of number of inputs

    int* r_h;
    hipHostMalloc((void**)&r_h, sizeof(int)*rSize); //use pinned memory in host so it copies to GPU faster
    dataGenerator(r_h, rSize, 0, 1); //randomly generate input data

    int* d_histogram; //histogram for algorithm 1
    hipMalloc((void**)&d_histogram, sizeof(int)*numP); //size of number of partitions

    int* h_histogram;
    hipHostMalloc((void**)&h_histogram, sizeof(int)*numP); //a bucket for each partition
    memset(h_histogram, 0, sizeof(int)*numP); //initialize host histogram to zero

    int* d_psum; //array to hold the prefix sum of algorithm 2
    hipMalloc((void**)&d_psum, sizeof(int)*numP); //a bucket for each partition

    int* h_psum;
    hipHostMalloc((void**)&h_psum, sizeof(int)*numP);

    int* d_output; //output array for final sorted result
    hipMalloc((void**)&d_output, sizeof(int)*rSize);

    int* h_output;
    hipHostMalloc((void**)&h_output, sizeof(int)*rSize);
        

    //copy host data to device memory
    hipMemcpy(r_d, r_h, sizeof(int)*rSize, hipMemcpyHostToDevice);
    hipMemcpy(d_histogram, h_histogram, sizeof(int)*numP, hipMemcpyHostToDevice);


    printf("The number of elements in the input array is: %d\n",rSize);
    printf("The number of partitions is: %d\n",numP);
    printf("The number of bits in a tag is: %d\n\n",tag);

    printf("The contents of the input array are: \n");
    for(int i = 0; i < rSize && i < 100; i++){
        
        printf("%d\n",r_h[i]);
    }
   

    //define block and grid size for algorithm 1 and 3. 2 only runs with 1 block
    int num_threads = 1024; //number of threads in a block
    int num_blocks = (rSize + num_threads - 1)/num_threads;

    //start counting time
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
    hipEventRecord(start, 0);
    
    //launch kernel 1 - create histogram
        histogram<<<1024, 256, numP*sizeof(int)>>>(r_d, d_histogram, tag, rSize, numP);

        //copy data from gpu to host
        hipMemcpy(h_histogram, d_histogram, sizeof(int)*numP, hipMemcpyDeviceToHost);

    //launch kernel 2 - exclusive prefix sum of histogram
        prefixscan<<<1, numP/2, numP*sizeof(int)>>>(d_histogram, d_psum, numP);

        //copy data from gpu to host
        hipMemcpy(h_psum, d_psum, sizeof(int)*numP, hipMemcpyDeviceToHost);

        /*//print psum
        printf("First 100 of exclusive prefix: \n");
        for(int i = 0; i < numP && i < 50; i++){
            printf("%d\n",h_psum[i]);
        }
        printf("\n");*/

    //launch kernel 3 - reorder input array
        Reorder<<<num_blocks, num_threads>>>(r_d, d_output, d_psum, tag, rSize);

        //copy final result from gpu to host
        hipMemcpy(h_output, d_output, sizeof(int)*rSize, hipMemcpyDeviceToHost);

    //stop counting time
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	float elapsedTime;
    hipEventElapsedTime(&elapsedTime, start, stop);

    //print sorted result
    printf("First 50 of the sorted output: \n");
    for(int i = 0; i < rSize && i < 50; i++){
    printf("%d\n",h_output[i]);
    }

    //print formatted output
    output_result(h_histogram, h_psum, numP);

    //report running time
	printf("******** Total Running Time of All Kernels = %0.5f ms *******\n", elapsedTime);
	hipEventDestroy(start);
	hipEventDestroy(stop);

    hipHostFree(r_h);
    hipHostFree(h_histogram);

    return 0;
}
