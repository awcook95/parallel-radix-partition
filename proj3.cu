#include "hip/hip_runtime.h"
#include <assert.h>
#include <stdio.h>
#include <math.h>

#define RAND_RANGE(N) ((double)rand()/((double)RAND_MAX + 1)*(N))

//data generator
void dataGenerator(int* data, int count, int first, int step)
{
	assert(data != NULL);

	for(int i = 0; i < count; ++i)
		data[i] = first + i * step;
	srand(time(NULL));
    for(int i = count-1; i>0; i--) //knuth shuffle
    {
        int j = RAND_RANGE(i);
        int k_tmp = data[i];
        data[i] = data[j];
        data[j] = k_tmp;
    }
}

//non random data generator
void nrdataGenerator(int* data, int count, int first, int step)
{
	assert(data != NULL);

	for(int i = 0; i < count; ++i)
		data[i] = first + i * step;
	//srand(time(NULL));
    for(int i = count-1; i>0; i--) //knuth shuffle
    {
        int j = RAND_RANGE(i);
        int k_tmp = data[i];
        data[i] = data[j];
        data[j] = k_tmp;
    }
}

/* This function embeds PTX code of CUDA to extract bit field from x. 
   "start" is the starting bit position relative to the LSB. 
   "nbits" is the bit field length.
   It returns the extracted bit field as an unsigned integer.
*/
__device__ uint bfe(uint x, uint start, uint nbits)
{
    uint bits;
    asm("bfe.u32 %0, %1, %2, %3;" : "=r"(bits) : "r"(x), "r"(start), "r"(nbits));
    return bits;
}

//Feel free to change the names of the kernels or define more kernels below if necessary

//define the histogram kernel here
__global__ void histogram(int* d_data, int* d_histogram, int tagLength, int size)
{
    int T = blockIdx.x * blockDim.x + threadIdx.x;

    if(T < size){
        int h = bfe(d_data[T], 0, tagLength);
        atomicAdd(&(d_histogram[h]), 1);
    }
}

//define the prefix scan kernel here
//implement it yourself or borrow the code from CUDA samples
__global__ void prefixscan(int *d_input, int *d_output, int n)
{
    extern __shared__ int shmem[];
    int T = threadIdx.x;
    int offset = 1;

    //there are n/2 threads so each thread must load 2 data points
    shmem[2*T] = d_input[2*T]; // load even indices into shared memory
    shmem[2*T+1] = d_input[2*T+1]; //load odd indices

    for (int d = n>>1; d > 0; d >>= 1) //upsweep, compute partial sums
    {
        __syncthreads();
        if (T < d)
        {
            int ai = offset*(2*T+1)-1;
            int bi = offset*(2*T+2)-1;
            shmem[bi] += shmem[ai];
        }
    offset *= 2;
    }

    if (T == 0) { shmem[n - 1] = 0; } //last element to 0
    for (int d = 1; d < n; d *= 2) //downsweep, use partial sums to complete the psum
    {
        offset >>= 1;
        __syncthreads();

        if (T < d){
         int ai = offset*(2*T+1)-1;
         int bi = offset*(2*T+2)-1;
         int temp = shmem[ai];
         shmem[ai] = shmem[bi];
         shmem[bi] += temp;
        }
    }

    __syncthreads();
    d_output[2*T] = shmem[2*T]; //write to global memory in even odd pairs like above
    d_output[2*T+1] = shmem[2*T+1];
}

//define the reorder kernel here
__global__ void Reorder(int* d_data, int* d_output, int* sum, int tagLength, int size)
{
    int T = blockIdx.x * blockDim.x + threadIdx.x;

    if(T < size){
        int h = bfe(d_data[T], 0, tagLength); //extract bits from input data
        int offset = atomicAdd(&(sum[h]), 1); //calculate offset
        d_output[offset] = d_data[T]; //use the offset to place input data into correct partition
    }
}

bool isPowerOfTwo(unsigned long x)
{
    return (x != 0) && ((x & (x - 1)) == 0);
}

void outputHistogram(int* histogram, int buckets){
    printf("Histogram:");
    for(int i = 0; i < buckets && i < 10; i++){
        printf("\n%02d: ", i);
        printf("%15lld ", histogram[i]);
    }
}

void output_result(int* histogram, int* psum, int num_buckets){
	int i; 
    long long total_cnt = 0;
    printf("Offset and number of keys per parition:");
	for(i=0; i< num_buckets; i++) {
		if(i%5 == 0) /* we print 5 buckets in a row */
			printf("\n%02d: ", i);
		printf("%8d: %7lld ", psum[i],histogram[i]);
		total_cnt += histogram[i];
	  	/* we also want to make sure the total distance count is correct */
		if(i == num_buckets - 1)	
			printf("\n T:%lld \n", total_cnt);
		else printf("| ");
	}
}

int main(int argc, char const *argv[])
{
    int rSize = atoi(argv[1]); //number of elements in input array
    int numP = atoi(argv[2]); //number of partitions that input will be sorted into

    //errors for incorrect inputs
    if(argc > 3){
		printf("Too many command line arguments, ending program\n");
		return 0;
	}

	else if(argc < 3){
		printf("Too few command line arguments, ending program\n");
		return 0;
    }
    
    if(rSize <= 0 || numP > 1024 || numP <=0){ //input size must be >= 0 and max # partitions is 1024
		printf("Invalid command line arguments, ending program\n");
		return 0;
	}
    
    int* r_h; //input array

    hipHostMalloc((void**)&r_h, sizeof(int)*rSize); //use pinned memory in host so it copies to GPU faster
    
    nrdataGenerator(r_h, rSize, 0, 1); //randomly generate input data
    

    assert(numP <= rSize && isPowerOfTwo(numP)); //number of partitions must be less than or equal to the input array size and power of 2

    int tag = int(log2(float(numP))); //define number of bits in a tag

    
    printf("The number of elements in the input array is: %d\n",rSize);
    printf("The number of partitions is: %d\n",numP);
    printf("The number of bits in a tag is: %d\n\n",tag);

    printf("The contents of the input array are: \n");
    for(int i = 0; i < rSize && i < 100; i++){
        
        printf("%d\n",r_h[i]);
    }

    //allocate memory for host

    //(input array already allocated above)

    int* h_histogram; //host histogram
    hipHostMalloc((void**)&h_histogram, sizeof(int)*numP); //a bucket for each partition

    memset(h_histogram, 0, sizeof(int)*numP); //initialize host histogram to zero

    //Allocate device memory
    int* r_d; //input array for device
    int* d_histogram; //histogram for device

    hipMalloc((void**)&r_d, sizeof(int)*rSize); //size of number of inputs
    hipMalloc((void**)&d_histogram, sizeof(int)*numP); //size of number of partitions

    //copy host data to device memory
    hipMemcpy(r_d, r_h, sizeof(int)*rSize, hipMemcpyHostToDevice);
    hipMemcpy(d_histogram, h_histogram, sizeof(int)*numP, hipMemcpyHostToDevice);

    //define block and grid size
    int num_threads = 32; //number of threads in a block
    int num_blocks = (rSize + num_threads - 1)/num_threads;

    //start counting time
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
    hipEventRecord(start, 0);
    
    //launch kernel 1 - create histogram
        histogram<<<num_blocks, num_threads>>>(r_d, d_histogram, tag, rSize);

        //copy data from gpu to host
        hipMemcpy(h_histogram, d_histogram, sizeof(int)*numP, hipMemcpyDeviceToHost);

        //print output
        outputHistogram(h_histogram, numP);
        printf("\n");

    //launch kernel 2 - exclusive prefix sum of histogram

        //create the prefix sum array for device and host
        int* d_psum;
        int* h_psum;
        hipMalloc((void**)&d_psum, sizeof(int)*numP); //a bucket for each partition
        hipHostMalloc((void**)&h_psum, sizeof(int)*numP); 

        prefixscan<<<1, numP/2, numP*sizeof(int)>>>(d_histogram, d_psum, numP);

        //copy data from gpu to host
        hipMemcpy(h_psum, d_psum, sizeof(int)*numP, hipMemcpyDeviceToHost);

        //print psum
        printf("First 100 of exclusive prefix: \n");
        for(int i = 0; i < numP && i < 50; i++){
            printf("%d\n",h_psum[i]);
        }
        printf("\n");

    //launch kernel 3 - reorder input array
        //create output arrays for device and host
        int* d_output;
        int* h_output;

        //allocate memory
        hipMalloc((void**)&d_output, sizeof(int)*rSize);
        hipHostMalloc((void**)&h_output, sizeof(int)*rSize);

        Reorder<<<num_blocks, num_threads>>>(r_d, d_output, d_psum, tag, rSize);

        //copy final result from gpu to host
        hipMemcpy(h_output, d_output, sizeof(int)*rSize, hipMemcpyDeviceToHost);

        //print sorted result
        printf("First 50 of the sorted output: \n");
        for(int i = 0; i < rSize && i < 50; i++){
        printf("%d\n",h_output[i]);
        }

    //stop counting time
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	float elapsedTime;
    hipEventElapsedTime(&elapsedTime, start, stop);

    //print formatted output
    output_result(h_histogram, h_psum, numP);

    //report running time
	printf("******** Total Running Time of All Kernels = %0.5f ms *******\n", elapsedTime);
	hipEventDestroy(start);
	hipEventDestroy(stop);

    hipHostFree(r_h);
    hipHostFree(h_histogram);

    return 0;
}
